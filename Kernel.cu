#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "lodepng.h"

#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <string>

using namespace std;

constexpr auto matSize = 16;

hipError_t imageRectificationWithCuda(string& inputImg, string& outImg, int& numOfThreads);

__global__ void imgRectificationKernel(int* matrix)
{
	int i = blockIdx.x * 4 + threadIdx.x;
	if (matrix[i] < 127) {
		matrix[i] = 127;
	}
}

int main(int argc, char* argv[])
{
	string inputImgName = "";
	string outImgName = "";
	int numOfThreads = 0;

	if (argc != 4 || argv[1] == NULL || argv[2] == NULL || argv[3] == NULL ||
		argv[1] == "-h" || argv[1] == "--help" || argv[1] == "--h") {
		cout << "Assignment1.exe <name of input png> <name of output png> < # threads>" << endl;
	}
	else {
		if (argv[1] != NULL) {
			inputImgName = argv[1];
		}
		if (argv[2] != NULL) {
			outImgName = argv[2];
		}
		if (argv[3] != NULL) {
			numOfThreads = stoi(argv[3]);
		}
	}

	std::cout << "Name of Input Image File: " << inputImgName << std::endl;
	std::cout << "Name of Output Image File: " << outImgName << std::endl;
	std::cout << "Name of Output Image File: " << numOfThreads << std::endl;

	hipError_t status = imageRectificationWithCuda(inputImgName, outImgName, numOfThreads);
	return 0;
}

hipError_t imageRectificationWithCuda(string &inputImg, string &outImg, int &threads)
{
	int sizeOfMat = 16;
	int matrix[4][4] = { {126, 128, 122, 100}, {129, 120, 350, 300}, {122, 135, 127, 129}, {140, 145, 190, 195} };

	int* dev_Mat = nullptr;


	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)& dev_Mat, 16 * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_Mat, &matrix, 16 * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy host to dev failed!");
		goto Error;
	}

	int numOfThreadsPerBlock = 4;
	// Launch kernel on the GPU with one thread for each element.
	imgRectificationKernel <<<(sizeOfMat + (numOfThreadsPerBlock - 1)) / numOfThreadsPerBlock, numOfThreadsPerBlock >> > (dev_Mat);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "imgRectificationKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching imgRectificationKernel!\n", cudaStatus);
		goto Error;
	}

	int* output[4][4] = { 0 };
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(output, dev_Mat, sizeOfMat * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	for (int i = 0; i < 16; i++) {
		printf("%i", output[i / 4][i % 4]);
	}

Error:
	hipFree(dev_Mat);

	return cudaStatus;
}

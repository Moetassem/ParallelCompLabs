#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "lodepng.h"

#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <string>
#include <vector>

#define LODEPNG_COMPILE_DECODER

using namespace std;
using namespace lodepng;

//constexpr auto matSize = 16;

hipError_t imageRectificationWithCuda(std::vector<unsigned char>& inputImage, int& numOfThreads, unsigned char* outputImgPtr);

__global__ void imgRectificationKernel(unsigned char* matrix)
{
	int i = blockIdx.x * 4 + threadIdx.x;
	if (matrix[i] < 127) {
		matrix[i] = 127;
	}
}

int main(int argc, char* argv[])
{
	string inputImgName = "";
	string outImgName = "";
	int numOfThreads = 0;

	if (argc != 5 || argv[1] == NULL || argv[2] == NULL || argv[3] == NULL || argv[4] == NULL ||
		argv[1] == "-h" || argv[1] == "--help" || argv[1] == "--h") {
		cout << "Assignment1.exe <Command> <name of input png> <name of output png> < # threads>" << endl;
	}
	else {
		if (argv[2] != NULL) {
			inputImgName = argv[2];
		}
		if (argv[3] != NULL) {
			outImgName = argv[3];
		}
		if (argv[4] != NULL) {
			numOfThreads = stoi(argv[4]);
		}
	}

	std::vector<unsigned char> inputImage;
	unsigned width = 0;
	unsigned height = 0;

	if (decode(inputImage, width, height, inputImgName) != 0) {
		cout << "You F**ed up decoding the image" << endl;
		return 0;
	}

	unsigned char* outputImgPtr = nullptr;

	if (argv[1] != NULL && argv[1] == "rectify") {
		hipError_t status = imageRectificationWithCuda(inputImage, numOfThreads, outputImgPtr);
		cout << status << endl;
	}

	//else if (argv[1] != NULL && argv[1] == "pool") {
		//TODO::pool function
	//}
	//scout << *outputImgPtr << endl;

	/*if (encode(outImgName, outputImgPtr, width, height, LodePNGColorType::LCT_RGBA, 8) != 0) {
		cout << "You f**ed up encoding the image" << endl;
		return 0;
	}*/
		
	std::cout << "Name of Input Image File: " << inputImgName << std::endl;
	std::cout << "Name of Output Image File: " << outImgName << std::endl;
	std::cout << "Name of Output Image File: " << numOfThreads << std::endl;

	//hipError_t status = imageRectificationWithCuda(inputImgName, outImgName, numOfThreads);
	return 0;
}

hipError_t imageRectificationWithCuda(vector<unsigned char> &inputImage, int& numOfThreads, unsigned char* outputImgPtr)
{
	int sizeOfMat = inputImage.size();

	unsigned char* dev_Mat = nullptr;


	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)& dev_Mat, sizeOfMat * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_Mat, &inputImage, sizeOfMat * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy host to dev failed!");
		goto Error;
	}

	int numOfThreadsPerBlock = 1024;
	// Launch kernel on the GPU with one thread for each element.
	imgRectificationKernel <<<(numOfThreads + (numOfThreadsPerBlock - 1)) / numOfThreadsPerBlock, (numOfThreads % numOfThreadsPerBlock) >> > (dev_Mat);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "imgRectificationKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching imgRectificationKernel!\n", cudaStatus);
		goto Error;
	}

	unsigned char* output = nullptr;
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(output, dev_Mat, sizeOfMat * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	outputImgPtr = output;
	cout << output << endl;

Error:
	hipFree(dev_Mat);

	return cudaStatus;
}
